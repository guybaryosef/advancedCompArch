#include "hip/hip_runtime.h"


#include <math.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>

#include "../utils/otherFunctions.cpp"

#define THREADS_PER_BLOCK 1024
#define BLOCKS_PER_GRID  2*65535


#include "cpuFunctions.h"


///////// FUNCTION DECLERATIONS /////////
std::vector<int> gpu_outerLoop(
    const   std::vector<int>    hx,         // input signal
    const   std::vector<int>    hfilter_t,  // time component of filter
    const   std::vector<int>    hfilter_f,  // freq component of filter
    const   unsigned            fs,         // length of filter
    const   unsigned            B,          // number of bins
    const   unsigned            B_t,        // 2k (k== number of non-zero freq coeff in output vector)
    const   unsigned            W,          // the length of some intermediate vectors
    const   unsigned            L,          // number of loops in outside loop
    const   unsigned            L_c,        // number of loops (times to execute) the locLargeCoef function
    const   unsigned            L_t,        // loop threshold for the revHash function
    const   unsigned            L_l)        // threshold of whether or not to implement the revHash function
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	// allocating and copying memory to GPU
	int input_size = hx.size()*sizeof(int);


	int *hx_ptr = new int[input_size];
	
	for (int i=0; i<hx.size(); ++i)
		hx_ptr[i] = hx[i];

	int *hfilter_t_ptr = new int[fs];
	int *hfilter_f_ptr = new int[fs];
	for (int i=0; i<fs; ++i)
	{
		hfilter_t_ptr[i] = hfilter_t[i];
		hfilter_f_ptr[i] = hfilter_f[i];
	}

	int *dx = nullptr;
	err = hipMalloc((void **)&dx, input_size);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector x (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dx, hx_ptr, input_size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to copy vector x from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }
  
	int *dfilter_t = nullptr;
	err = hipMalloc((void **)&dfilter_t, fs*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector filter_t (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(dfilter_t, hfilter_t_ptr, fs*sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to copy vector filter_t from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }
	
	int *dfilter_f = nullptr;
	err = hipMalloc((void **)&dfilter_f, fs*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector filter_f (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(dfilter_f, hfilter_f_ptr, fs*sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to copy vector filter_f from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }

	int *dbins_t = nullptr;
	err = hipMalloc((void **)&dbins_t, B*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector bins_t_x (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }

	int *dbins_f = nullptr;
	err = hipMalloc((void **)&dbins_f, B*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector bins_f (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }
    
	int *dI = nullptr;
	err = hipMalloc((void **)&dI, input_size);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector I (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }

	int *dJ2 = nullptr;
	err = hipMalloc((void **)&dJ2, B_t*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector J_2 (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }

	int *dH_sig = nullptr;
	err = hipMallocManaged((void **)&dH_sig, L*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector H_sig (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
    }

	// executing the algorithm
    for (int i=0; i<L_c; ++i)
        gpu_locLargeCoefGPU(dx, B_t, hx.size(), W, dJ2);

	int *IF;
	err = hipMallocManaged(&IF, 2*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate unified memory integer IF (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	IF[0] = 0;
	
    for (int i=0; i<L; ++i)
    {
		int sigma = rand() % hx.size();
		std::cout << "i: " << i << std::endl;
        dH_sig[i] = modInverse(sigma, hx.size());

        int *dJ = nullptr;
		err = hipMalloc((void **)&dJ, B_t*sizeof(int));
		if (err != hipSuccess)
		{
			std::cerr << "Failed to allocate device vector J (error code " << hipGetErrorString(err) << ")!" << std::endl;
			exit(EXIT_FAILURE);
		}

        gpu_permFilter(dx, hx.size(), dfilter_t, fs, dbins_t, B, dH_sig[i]);
        gpu_fftCutoff(dJ, dbins_t, dbins_f, B, B_t);

        if (i < L_l)
				gpu_revHash(dI, dJ, B_t, B, hx.size(), L_t, dJ2, W, IF, sigma);
				
		err = hipFree(dJ);
		if (err != hipSuccess)
		{
			std::cerr << "Failed to free device vector J (error code " << hipGetErrorString(err) << ")!" << std::endl;
			exit(EXIT_FAILURE);
		}
    }

    int *output = gpu_eval(dI, IF, dbins_f, dfilter_f, B, hx.size(), L, dH_sig);

	// freeing all the GPU memory
	err = hipFree(dx);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector x (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
    
	err = hipFree(dbins_t);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector bins_t_x (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(dbins_f);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector bins_f (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(dI);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector I (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(dJ2);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector J2 (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(dH_sig);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free uniform memory vector H_sig (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(IF);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free uniform memory integer IF (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(dfilter_f);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector filter_f (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(dfilter_t);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector filter_t (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

    return std::vector<int>(output, output+IF[0]);
}


void gpu_locLargeCoefGPU(
	const   int                *dx,         // input signal
	const   unsigned            B_t,        // 2k (k== number of non-zero freq coeffs in output vector)
	const   unsigned            n,          // the length of the input vector
	const   unsigned            W,          // the length of the output vector
			int                *dJ2)        // function output
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	int *dx_prime;
	err = hipMalloc((void **)&dx_prime, W*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector x_prime (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	int *dy_hat = nullptr;
	err = hipMalloc((void **)&dy_hat, W*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector y_hat (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	int sigma = n/W;
	int tau   = rand() % sigma;
	LLC_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dx_prime, dx, W, tau, sigma);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch LLC_kernel (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

	gpu_fftCutoff(dJ2, dx_prime, dy_hat, W, B_t);

	err = hipFree(dy_hat);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector y_hat (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(dx_prime);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector x_prime (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
}


void gpu_fftCutoff(
			int				   *dvec,      	// input vector in freq domain after cutoff
	const   int			   	   *dinvec_t,  	// input vector in the time domain
			int				   *dinvec_f,  	// input vector in the freq domain
	const   int       			n_bins,   	// number of bins (length of input vector in both time & freq)
	const   unsigned            B_t)        // 2k (length of dvec)
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	// executing cuFFT
	int dft_length= n_bins/2 + 1; 
	int dft_batch = 1;	// number of dfts

	hipfftComplex *dft_output;
	err = hipMalloc((void**)&dft_output, sizeof(hipfftComplex)*dft_length);
	if (hipGetLastError() != hipSuccess)
	{
		std::cerr << "Cuda error: Failed to allocate dft output for cuFFT." << std::endl;
		return;	
	}

	hipfftReal *dft_input;
	err = hipMalloc((void**)&dft_input, sizeof(hipfftReal)*n_bins);
	if (hipGetLastError() != hipSuccess)
	{
		std::cerr << "Cuda error: Failed to allocate dft input for cuFFT." << std::endl;
		return;	
	}

	intToCureal_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dft_input, dinvec_t, n_bins);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch intToCureal_kernel (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

	hipfftHandle plan;
	if (hipfftPlan1d(&plan, n_bins, HIPFFT_R2C, dft_batch) != HIPFFT_SUCCESS)
	{
		std::cerr << "CUFFT error: Plan creation failed" << std::endl;
		return;	
	}	

	if (hipfftExecR2C(plan, dft_input, dft_output) != HIPFFT_SUCCESS)
	{
		std::cerr << "CUFFT error: ExecC2C Forward failed." << std::endl;
		return;	
	}

	if (hipDeviceSynchronize() != hipSuccess)
	{
		std::cerr << "Cuda error: Failed to synchronize." << std::endl;
		return;	
	}

	curealToInt_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dft_output, dinvec_f, n_bins);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch curealToInt (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

	err = hipFree(dft_output);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector dft_output (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(dft_input);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector dft_input (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipfftDestroy(plan);

	gpu_cutOff(dvec, B_t, dinvec_f, n_bins);
}


void gpu_cutOff(
			int                *dId,        // the B_t largest freq coeffs in the bins vector in freq domain
	const   unsigned            B_t,        // 2k (length of dId)
			int                *d_yhat,     // vector in freq domain
	const   unsigned            m)          // the length of d_yhat
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	int *dsamples_s = nullptr;
	err = hipMalloc((void **)&dsamples_s, m*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector samples_s (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	int *dsamples_I = nullptr;
	err = hipMalloc((void **)&dsamples_I, m*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector samples_I (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	S_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_yhat, dsamples_s, dsamples_I, m);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch S_kernel (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

    thrust::device_ptr<int> dsamples_s_thrust = thrust::device_pointer_cast(dsamples_s);
	thrust::sort(dsamples_s_thrust, dsamples_s_thrust+m);
	dsamples_s = thrust::raw_pointer_cast(dsamples_s_thrust);

	int cutoff = dsamples_s_thrust[m-B_t-1];


	int *id = nullptr;
	err = hipMalloc((void **)&id, sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device int id (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	C_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dId, cutoff, dsamples_I, m, id, B_t);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch C_kernel (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

	thrust::device_ptr<int> dId_thrust = thrust::device_pointer_cast(dId);
	thrust::sort(dId_thrust, dId_thrust+B_t);
	dId = thrust::raw_pointer_cast(dId_thrust);

	err = hipFree(dsamples_s);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector samples_s (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(dsamples_I);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector samples_I (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
}


void gpu_permFilter(
	const   int   			   *dx,         // input signal
	const	int					n,			// length of the input signal
	const   int				   *dfilt_t,    // time component of filter
	const	int					fs,			// the length of the filter
			int				   *dbins_t,    // permuted and filtered components of input signal are binned here
	const   unsigned            B,          // number of bins
	const   int                 dH_sig_i)    // modular inverse of sigma and n
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	err = hipMemset(dbins_t, 0, B);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to set memory of device vector bins_t (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	int T = fs / B;
	int R = fs % B;

	if (n < pow(2, 27))
	{
		PFT_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dbins_t, dx, dfilt_t, n, B, dH_sig_i, T, R);
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "Failed to launch PFT_kernel (error code " << hipGetErrorString(err) << ")!\n";
			exit(EXIT_FAILURE);
		}
	}
	else
	{
		PFK_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dbins_t, dx, dfilt_t, n, B, dH_sig_i, fs);
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "Failed to launch PFK_kernel (error code " << hipGetErrorString(err) << ")!\n";
			exit(EXIT_FAILURE);
		}
	}
}


void gpu_revHash(
				int			   *dI, 
		const	int			   *dJ, 
		const	unsigned		B_t, 
		const	unsigned		B, 
		const	unsigned		n, 
		const	unsigned		L_t, 
		const	int			   *dJ2, 
		const	unsigned		W, 
				int			   *IF, 
		const	int				sigma)
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	int *dV = nullptr;
	err = hipMalloc((void **)&dV, n*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector V (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}	

	err = hipMemset(dV, 0, n);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to set memory of device vector V (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	int *dJ_2sig = nullptr;
	err = hipMalloc((void **)&dJ_2sig, B_t*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector J_2sig (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}	

	for (int i=0; i<B_t; ++i)
		dJ_2sig[i] = (dJ2[i]*sigma) % W;

	RH_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dI, dJ, dV, dJ_2sig, L_t, IF, B_t);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch RH_kernel (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

	err = hipFree(dV);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector V (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	err = hipFree(dJ_2sig);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to free device vector J_2sig (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}
}


int *gpu_eval(
		const	int			   *dI, 
		const	int			   *IF, 
		const	int			   *dbins_f, 
		const	int			   *dfilter_f, 
		const	unsigned		B, 
		const	unsigned		n, 
		const	unsigned		L, 
		const	int			   *dH_sig)
{
	hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

	int *dx_hat = nullptr;
	err = hipMalloc((void **)&dx_hat, n*sizeof(int));
	if (err != hipSuccess)
	{
		std::cerr << "Failed to allocate device vector x_hat (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	EV_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dx_hat, dI, IF, dbins_f, L, n, dH_sig, B, dfilter_f);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Failed to launch EV_kernel (error code " << hipGetErrorString(err) << ")!\n";
		exit(EXIT_FAILURE);
	}

	int *hx_hat = new int[IF[0]];
	err = hipMemcpy(hx_hat, dx_hat, IF[0]*sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		std::cerr << "Failed to copy vector x_hat from device to host (error code " << hipGetErrorString(err) << ")!" << std::endl;
		exit(EXIT_FAILURE);
	}

	return hx_hat;
}
